#include "hip/hip_runtime.h"
#ifndef cudaEn
	#define cudaEn
#endif

//Shared Memory 
#include "ShMemSymBuff_cucomplex.hpp"
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <assert.h>
#define FFT_size dimension
#define cp_size prefix
#define numSymbols lenOfBuffer

//gpu

#define threadsPerBlock FFT_size
#define numOfBlocks numOfRows

//LS
#define fileNameForX "Pilots.dat"
#define mode 0
/*
	mode:
		= 1 -> master -> creates shared memory 
		= 0 -> slave -> doesn't create the shared memory
*/
 
//!How to Compile:   nvcc ../../examples/gpuLS_cucomplex.cu -lcufft -lrt -o gpu 
// ./gpu

//LS
//Y = 16 x 1024
//X = 1 x 1023
//H = 16 x 1023
ShMemSymBuff* buffPtr;

using namespace std;

std::string file = "Output_gpu.dat";
//std::ofstream outfile;
	
	
//Reads in Vector X from file -> 1xcols
void matrix_readX(hipFloatComplex* X, int cols){
	ifstream inFile;
	inFile.open(fileNameForX);
	if (!inFile) {
		cerr << "Unable to open file "<< fileNameForX<<", filling in 1+i for x\n";
		float c=1.0f;
		for (int col = 0; col <  cols; col++){
			X[col].x=c;
			X[col].y=c;
		}
		return;
	}
	inFile.read((char*)X, (cols)*sizeof(*X));
	/*
	float c=0;
	for (int col = 0; col <  cols; col++){
		inFile >> c;
		X[col].real=c;
		inFile >> c;
		X[col].imag=c;
	}
	*/
	hipFloatComplex* temp = 0;
	temp=(hipFloatComplex*)malloc ((cols-1)/2* sizeof (*temp));
	//copy second half to temp
	memmove(temp, &X[(cols+1)/2], (cols-1)/2* sizeof (*X));
	//copy first half to second half
	memmove(&X[(cols-1)/2], X, (cols+1)/2* sizeof (*X));
	//copy temp to first half
	memmove(X, temp, (cols-1)/2* sizeof (*X));
	
	free(temp);
	inFile.close();
}

void shiftOneRow(hipFloatComplex* Y, int cols, int row){
	hipFloatComplex* Yf = &Y[row*cols];
	//std::cout << "Here...\n";
	hipFloatComplex* temp = 0;
	temp=(hipFloatComplex*)malloc ((cols+1)/2* sizeof (*temp));
	//copy second half to temp
	memmove(temp, &Yf[(cols-1)/2], (cols+1)/2* sizeof (*Yf));
	//copy first half to second half
	memmove(&Yf[(cols+1)/2], Yf, (cols-1)/2* sizeof (*Yf));
	//copy temp to first half
	memmove(Yf, temp, (cols+1)/2* sizeof (*Yf));
	
	free(temp);
	
}

void dropPrefix(hipFloatComplex *Y, hipFloatComplex *dY, int rows1, int cols1){
	
	int rows = rows1;
	int cols= cols1;	
	for(int i =0; i<rows; i++){
		memcpy(&Y[i*cols], &dY[i*(cols+prefix)+ prefix], cols*sizeof(*dY));
	}		
	
}

__global__ void findHs(hipFloatComplex* dY,hipFloatComplex* dH,hipFloatComplex* dX,int rows1,int cols1){
	
	//int rows = rows1;
	int cols=cols1;
	int rows = rows1;
	int row = blockIdx.x;
	int j = threadIdx.x;
	//find my work
	//Drop first element and copy it into Hconj
	/*
	__shared__ hipFloatComplex temp[threadsPerBlock-1];
	temp[j] = dY[row*(blockDim.x+1) + j + 1];
	__syncthreads();
	dH[row*blockDim.x + j] = temp[j];
	__syncthreads();
	*/
	dH[row*blockDim.x + j] = dY[row*(blockDim.x+1) + j + 1];
	__syncthreads();
	
	//complex division
	//H/X where H = FFT(Y) (w/ dropped first element)
	//Then take conjugate of H
	int i = blockIdx.x;
	//int j = threadIdx.x;
	//for(int j=0; j<c; j++){
	if (j < (cols-1)) {
		//dH[i*blockDim.x + j] = dY[i*blockDim.x + j + 1];
		dH[row*blockDim.x + j] = hipCdivf(dH[row*blockDim.x + j], dX[row*blockDim.x + j]);
		dH[row*blockDim.x + j] = hipConjf(dH[row*blockDim.x + j]);
	}
	//}
	__syncthreads();
	//Now dH holds conj H
}

void findDistSqrd(hipFloatComplex* H, float* Hsqrd, int rows, int cols){
	//initialize first row since Hsqrd currently holds X
	for (int j = 0; j<cols; j++){
		Hsqrd[j] = 0;
		//|H|^2 = real^2 + imag^2
		//Sum of |H|^2 is summing all elements in col j
		Hsqrd[j] = H[j].x*H[j].x + H[j].y*H[j].y;
		//Hsqrd[j].y = 0;
	}
	
	for (int i = 1; i<rows; i++){  
		for (int j = 0; j<cols; j++){
			//|H|^2 = real^2 + imag^2
			//Sum of |H|^2 is summing all elements in col j
			Hsqrd[j] = Hsqrd[j] + (H[i*cols + j].x*H[i*cols + j].x + H[i*cols + j].y*H[i*cols + j].y);
		}
	}
	
}

void firstVector(hipFloatComplex* dY, hipFloatComplex* dH, hipFloatComplex* dX, float* Hsqrd, int rows, int cols){
	clock_t start, finish;
	//std::cout << "Here...\n";
	//X = 1x1023 -> later can become |H|^2
	hipFloatComplex* X = 0;
	int sizeX=rows*(cols-1)* sizeof(*X);
	X = (hipFloatComplex*)malloc(sizeX);
	//complexF* H =0;
	//H = (complexF *)malloc(sizeX*rows);
	//hipMalloc((void**)&H, size);
	
	//Read in X vector -> 1x1023
	for (int i = 0; i < rows; i++) {
		//std::cout << "Here...\n";
		matrix_readX(&X[i*(cols-1)], cols-1);
	}
	//std::cout << "Here...\n";
	hipMemcpy(dX, X, rows*(cols-1)*sizeof(*dX), hipMemcpyHostToDevice);
	hipDeviceSynchronize();	
	
	// CUFFT plan -> do it one time before?
	
	
	//Read in Y with prefix
	buffPtr->readNextSymbol(dY, 0);
	decode[0]=0;
	//drop the prefix and move into first part of dY
	hipFloatComplex* Y = 0;
	hipMalloc((void**)&Y, rows*cols*sizeof(*Y));
	hipfftHandle plan;
    hipfftPlan1d(&plan, cols, HIPFFT_C2C, rows);
	hipfftExecC2C(plan, (hipfftComplex *)Y, (hipfftComplex *)Y, HIPFFT_FORWARD);
	if(timerEn){
		start = clock();
	}
	hipMemcpy(Y, dY, rows*cols*sizeof(*Y), hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	if(timerEn){
		finish = clock();
		readT[0] = readT[0] + ((float)(finish - start))/(float)CLOCKS_PER_SEC;
	}
//	std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
	
	/*
	if(prefix>0){
		clock_t start, finish;
		if(timerEn){
			start = clock();
		}
		dropPrefix(Y, dY, rows, cols);
		hipDeviceSynchronize();
		if(timerEn){
			finish = clock();
			drop[0] = ((float)(finish - start))/(float)CLOCKS_PER_SEC;
		}
	}
	*/
	
		//hipfftExecC2C(plan, (hipfftComplex *)dY, (hipfftComplex *)dY, HIPFFT_FORWARD);
	
	if(timerEn){
		start = clock();
	}
	
	
	//FFT(Y)
//	hipfftHandle plan;
//	hipfftPlan1d(&plan, cols, HIPFFT_C2C, rows);
	hipfftExecC2C(plan, (hipfftComplex *)Y, (hipfftComplex *)Y, HIPFFT_FORWARD);
	hipDeviceSynchronize();
	if(timerEn){
		finish = clock();
		fft[0] = fft[0]+ ((float)(finish - start))/(float)CLOCKS_PER_SEC;
	}
	
	//find Hconj and Hsqrd
	if(timerEn){
		start = clock();
	}
	findHs<< <numOfBlocks,threadsPerBlock-1 >> >(Y, dH, dX, rows, cols);
	hipDeviceSynchronize();
	/*
	if(timerEn){
		finish = clock();
		decode[0] = decode[0]+ ((float)(finish - start))/(float)CLOCKS_PER_SEC;
	}
	*/
	//std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
	hipFloatComplex *H = 0;
	H = (hipFloatComplex*)malloc(rows*(cols-1)*sizeof(*H));
	hipMemcpy(H, dH, rows*(cols-1)*sizeof(*dH), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	//std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
	
	//H holds Hconj
	//hipMemcpy(H, dH, sizeX*rows, hipMemcpyDeviceToHost);
	/*
	if(timerEn){
		start = clock();
	}
	*/
	//Save |H|^2 into Hsqrd
	findDistSqrd(H,Hsqrd,rows, cols-1);
	
	
	if(timerEn){
		finish = clock();
		decode[0] = decode[0]+ ((float)(finish - start))/(float)CLOCKS_PER_SEC;
	}
	/*
	std::string file = "Chan_est.dat";
	hipFloatComplex* Yf;
	Yf = (hipFloatComplex*)malloc(rows*(cols-1)*sizeof(*Yf));
	hipMemcpy(Yf, dH, rows*(cols-1)*sizeof(*dH), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
	std::cout << "After Chan Est...\n";
	outfile.open(file.c_str(), std::ofstream::binary);
	outfile.write((const char*)Yf, rows*(cols-1)*sizeof(*Yf));
	outfile.close();
	*/
	/*
	memcpy(Yf, Hsqrd, (cols-1)*sizeof(*Hsqrd));
	std::cout << "After Squared...\n";
	file = "Dist_sqrd.dat";
	outfile.open(file.c_str(), std::ofstream::binary);
	outfile.write((const char*)Yf, (cols-1)*sizeof(*Yf));
	outfile.close();
	*/
	
	free(X);
	free(H);
	
	//dH holds H conj
	//dX holds {H^2)	
}


__global__ void doOneSymbol(hipFloatComplex* Y, hipFloatComplex* Hconj, hipFloatComplex* Yf, int rows1, int cols1){
	int rows = rows1;
	int cols= cols1;
    
    //find my work 
	//Y x conj(H) -> then sum all rows into elements in Hsqrd
	//Y = 16x1024+prefix
	//conjH = 16x1023
	int row = blockIdx.x;
	/*
	if (row == 0) {
		printf("%d\n",gridDim.x);
	}
	__syncthreads();
	*/
	int j = threadIdx.x;
	
//	for (int i = 0; i < rows; i++) {
		
//		for (int j = 0; j < cols-1; j++) {
	/*
	__shared__ hipFloatComplex temp[threadsPerBlock-1];
	temp[j] = Y[row*(blockDim.x+1) + j + 1];
	__syncthreads();
	Yf[row*blockDim.x + j] = temp[j];
	__syncthreads();
	*/
	Yf[row*blockDim.x + j] = Y[row*(blockDim.x+1) + j + 1];
	__syncthreads();
//		}
		
		//memcpy(&Yf[i*(cols-1)], &Y[i*cols+1], (cols-1)* sizeof (*Yf));
//	}
	
	if (j < cols-1) {
	//	Yf[i*c + j] = Y[i*c + j + 1];
		Yf[row*blockDim.x + j] = hipCmulf(Yf[row*blockDim.x + j],Hconj[row*blockDim.x + j]);
	}
	__syncthreads();
	//free(temp);
}

void symbolPreProcess(hipFloatComplex *Y, hipFloatComplex *Hconj, float *Hsqrd,int rows1, int cols1, int it) {
	int rows = rows1;
	int cols= cols1;

	clock_t start, finish;
	//Y x conj(H) -> then sum all rows into elements in Hsqrd
	//Y = 16x1024+prefix
	//conjH = 16x1023
	if(timerEn){
		start = clock();
	}
	
	hipFloatComplex* dY = 0;
	hipMalloc((void**)&dY, rows*cols*sizeof(*dY));
	hipMemcpy(dY, Y, rows*cols*sizeof(*dY), hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	
	if(timerEn){
		finish = clock();
		readT[it] = readT[it] + ((float)(finish - start))/(float)CLOCKS_PER_SEC;
	}
//	std::cout << "Symbol " << it << ": " << hipGetErrorString(hipGetLastError()) << std::endl;
	

	/*
	if (it == 1) {
		std::string file = "Prefix_drop.dat";
		hipFloatComplex *Yf;
		Yf = (hipFloatComplex*)malloc(rows*cols*sizeof(*Yf));
		hipMemcpy(Yf, dY, rows*cols*sizeof(*Yf), hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
		std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
		std::cout << "\n After Prefix drop:\n";
		for (int j = 0; j < rows*(cols); j = j + cols) {
			cout << "(" << Yf[j].x << ", " << Yf[j].y << ")\n";
		}
		
		outfile.open(file.c_str(), std::ofstream::binary);
		outfile.write((const char*)Yf, rows*(cols)*sizeof(*Yf));
		outfile.close();
		free(Yf);
	}
	*/
	
	if(timerEn){
		start = clock();
	}
	
	//FFT(Y)
	hipfftHandle plan;
    hipfftPlan1d(&plan, cols, HIPFFT_C2C, rows);
	hipfftExecC2C(plan, (hipfftComplex *)dY, (hipfftComplex *)dY, HIPFFT_FORWARD);
	hipDeviceSynchronize();
	if(timerEn){
		finish = clock();
		fft[it] = fft[it]+ ((float)(finish - start))/(float)CLOCKS_PER_SEC;
	}
	
	/*
	if (it == 1) {
		std::string file = "FFT_Out.dat";
		hipFloatComplex* Yf;
		Yf = (hipFloatComplex*)malloc(rows*cols*sizeof(*Yf));
		hipMemcpy(Yf, dY, rows*cols*sizeof(*Yf), hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
		std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
		std::cout << "After FFT...\n";
		outfile.open(file.c_str(), std::ofstream::binary);
		outfile.write((const char*)Yf, rows*(cols)*sizeof(*Yf));
		outfile.close();
		free(Yf);
	}
	*/
	
	hipFloatComplex* Yf = 0;
	hipMalloc((void**)&Yf, rows*(cols-1)* sizeof (*Yf));
	
	if(timerEn){
		start = clock();
	}
	doOneSymbol<< <numOfBlocks,(threadsPerBlock-1)>> >(dY, Hconj, Yf, rows, cols);
	hipDeviceSynchronize();
	hipMemcpy(Y, Yf, rows*(cols-1)*sizeof(*Y), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	/*
	if(timerEn){
		start = clock();
	}
	*/
	for(int r=1; r<rows; r++){
		for(int j=0; j<cols-1; j++){
			Y[j]= hipCaddf(Y[j],Y[r*(cols-1)+j]);
		}
	}
	
	//Divide YH* / |H|^2
	for(int j=0; j<cols-1; j++){
		Y[j].x = Y[j].x/Hsqrd[j];
		Y[j].y = Y[j].y/Hsqrd[j];
	}
	
	shiftOneRow(Y, cols-1, 0);
	
	if(timerEn){
		finish = clock();
		decode[it] = ((float)(finish - start))/(float)CLOCKS_PER_SEC;
	}
	hipFree(dY);
	hipFree(Yf);
	hipDeviceSynchronize();
}



int main(){
	int rows = numOfRows; // number of vectors
	int cols=dimension;//dimension
	hipSetDevice(0);
	//printf("CUDA LS: \n");
	//printInfo();
	//dY holds symbol with prefix
	hipFloatComplex *dY = 0;
	dY = (hipFloatComplex*)malloc(rows*cols* sizeof (*dY));
	
	float *Hsqrd = 0;
	Hsqrd = (float*)malloc((cols-1)*sizeof (*Hsqrd));
	
	//dH (and Hconj) = 16x1023
	hipFloatComplex *dH = 0;
	hipMalloc((void**)&dH, rows*(cols-1)* sizeof (*dH));
	
	//X = 1x1023 -> later can become |H|^2
	hipFloatComplex *dX = 0;
	hipMalloc((void**)&dX, rows*(cols-1)* sizeof (*dX));
	
	hipFloatComplex *Yf = 0;
	Yf = (hipFloatComplex*)malloc((cols-1)* sizeof (*Yf));
	
	//Shared Memory
	string shm_uid = shmemID;
	buffPtr=new ShMemSymBuff(shm_uid, mode);
	
	
	/*hipfftComplex *temp = 0;
	hipMalloc((void**)&temp, rows*cols* sizeof (*temp));
	hipfftHandle plan;
    hipfftPlan1d(&plan, cols, HIPFFT_C2C, rows);
	hipfftExecC2C(plan, (hipfftComplex *)&temp, (hipfftComplex *)&temp, HIPFFT_FORWARD);
	hipDeviceSynchronize();
	hipFree(temp);
	*/
	//Find H* (H conjugate) ->16x1023 and |H|^2 -> 1x1023
	firstVector(dY, dH, dX, Hsqrd, rows, cols);
	//dH holds h conj
	//dX holds |H|^2
	
	for(int i=1; i<numberOfSymbolsToTest; i++){
		if(i==numberOfSymbolsToTest-1){
			//if last one
			buffPtr->readLastSymbol(dY);
		}
		else{
			buffPtr->readNextSymbol(dY,i);
			/*
			if (i == 1) {
				std::string file = "Sym_copy.dat";
//				hipFloatComplex Yf_[rows*(cols+prefix)];
				hipFloatComplex *Yf_;
				Yf_ = (hipFloatComplex*)malloc(rows*(cols)*sizeof(*Yf_));
				memcpy(Yf_, dY, rows*(cols)*sizeof(*Yf_));
				//hipDeviceSynchronize();
				//std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
				std::cout << "Copied back to CPU...\n";
				//printOutArr(Yf_,1,cols+prefix);
					for (int j = 0; j < rows*(cols); j = j + cols) {
						cout << "(" << Yf_[j].x << ", " << Yf_[j].y << ")\n";
					}
				outfile.open(file.c_str(), std::ofstream::binary);
				outfile.write((const char*)Yf_, rows*(cols)*sizeof(*Yf_));
				outfile.close();
			}
			*/
			
		}
		symbolPreProcess(dY, dH, Hsqrd, rows, cols, i);
		
		if(testEn){
			//printf("Symbol #%d:\n", i);
			//cuda copy it over
			memcpy(Yf, dY, (cols-1)* sizeof (*Yf));
			if (i <= 1) {
				outfile.open(file.c_str(), std::ofstream::binary | std::ofstream::trunc);
			} else {
				outfile.open(file.c_str(), std::ofstream::binary | std::ofstream::app);
			}
			outfile.write((const char*)Yf, (cols-1)*sizeof(*Yf));
			outfile.close();
			//printOutArr(Yf, 1, cols-1);
		}
		
		
	}
	
	free(Yf);
	hipFree(dY);
	hipFree(dH);
	hipFree(dX);
	//delete buffPtr;
	
	if(timerEn) {
	//	printTimes(true);
		storeTimes(false);
	}
	return 0;

}